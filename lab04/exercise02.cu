#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define N 2050
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int *a);
void vectorAddCPU(int *a, int *b, int *c_ref);
int validate(int *a, int *c_ref);



__global__ void vectorAdd(int *a, int *b, int *c, int max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < max)
	  c[i] = a[i] + b[i];
}



int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	// Launch add() kernel on GPU
  // account for when threadsperblock does not divide N evenly = add extra block
  dim3 blocksPerGrid((int)ceil(N / (double)THREADS_PER_BLOCK), 1, 1);
  dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
	// vectorAdd << <N / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK >> >(d_a, d_b, d_c, N);
  vectorAdd <<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
	checkCUDAError("CUDA kernel");


	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

  //CPU version
  vectorAddCPU(a, b, c_ref);

  //validate
  int numErrors = validate(c, c_ref);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for (unsigned int i = 0; i < N; i++){
		a[i] = rand();
	}
}

void vectorAddCPU(int *a, int *b, int *c_ref) {
  
  for (int i = 0; i < N; i+=1) {
    c_ref[i] = a[i] + b[i];
  }
}

int validate(int *a, int *c_ref) {
  int errors = 0;

  for (int i = 0; i < N; i++) {
    if (a[i] != c_ref[i]) {
      errors += 1;
      fprintf(stderr, "Error at index %d: The result %d from vectorAdd does not equal the result %d from vectorAddCPU \n", i, a[i], c_ref[i]);
      printf("Total errors so far: %d \n", errors);
    }
  }

  printf("Total num errors: %d \n", errors);
  return errors;
}
